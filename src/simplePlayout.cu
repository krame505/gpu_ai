#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>

#define CUDA_STACK_SIZE 1024 * 32

#define SEED 12345

#define BLOCK_SIZE 32

__global__ void simplePlayoutKernel(State *states, PlayerId *results, int n) {
  uint8_t tx = threadIdx.x;
  uint32_t bx = blockIdx.x;
  uint32_t tid = tx + (bx * NUM_LOCS);

  if (tid < n) {
    State state;
    state = states[tid];

    // Init random generator
    hiprandState_t generator;
    hiprand_init(SEED, tid, 0, &generator);
 
    bool gameOver = false;

    Move captureMoves[MAX_LOC_MOVES * 12];
    Move directMoves[MAX_LOC_MOVES * 12];
    uint8_t numMoveCapture, numMoveDirect;

    do {
      // Scan the board for pieces that can move
      numMoveCapture = 0;
      numMoveDirect = 0;
      for (uint8_t i = 0; i < BOARD_SIZE; i++) {
	for (uint8_t j = 1 - (i % 2); j < BOARD_SIZE; j++) {
	  Loc here(i, j);
	  numMoveCapture += state.genLocCaptureMoves(here, &captureMoves[numMoveCapture]);
	  if (numMoveCapture == 0)
	    numMoveDirect += state.genLocDirectMoves(here, &directMoves[numMoveDirect]);
	}
      }

      if (numMoveCapture > 0) {
	state.move(captureMoves[hiprand(&generator) % numMoveCapture]);
      }
      else if (numMoveDirect > 0) {
	state.move(directMoves[hiprand(&generator) % numMoveDirect]);
      }
      else {
	gameOver = true;
      }
    } while (!gameOver);

    results[tid] = state.getNextTurn();
  }
}

std::vector<PlayerId> DeviceSimplePlayoutDriver::runPlayouts(std::vector<State> states) const {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));

  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);

  hipError_t error = hipDeviceSetLimit(hipLimitStackSize, CUDA_STACK_SIZE);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error setting stack size: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  int numBlocks = states.size() / BLOCK_SIZE;
  if (states.size() % BLOCK_SIZE)
    numBlocks++;
  
  // Invoke the kernel
  simplePlayoutKernel<<<numBlocks, BLOCK_SIZE>>>(devStates, devResults, states.size());
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
