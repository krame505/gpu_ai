#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>

#define CUDA_STACK_SIZE 1024 * 32

#define SEED 12345

__global__ void simplePlayoutKernel(State *states, PlayerId *results) {
  uint8_t tx = threadIdx.x;
  uint32_t bx = blockIdx.x;
  uint32_t tid = tx + (bx * NUM_LOCS);

  State state;
  state = states[tid];

  // Init random generator
  hiprandState_t generator;
  hiprand_init(SEED, tid, 0, &generator);
 
  bool gameOver = false;

  Loc playerOccupied[12];
  uint8_t numOccupied;
  
  do {
    numOccupied = 0;
    for (uint8_t i = 0; i < BOARD_SIZE; i++) {
      for (uint8_t j = 0; j < BOARD_SIZE; j++) {
	Loc here(i, j);
	if (state[here].occupied && state[here].owner == state.turn) {
	  playerOccupied[numOccupied++] = here;
	}
      }
    }
  } while (!gameOver);

  results[tid] = state.getNextTurn();
}

std::vector<PlayerId> DeviceSimplePlayoutDriver::runPlayouts(std::vector<State> states) const {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));

  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);

  hipError_t error = hipDeviceSetLimit(hipLimitStackSize, CUDA_STACK_SIZE);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error setting stack size: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Invoke the kernel
  simplePlayoutKernel<<<states.size(), NUM_LOCS>>>(devStates, devResults);
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
