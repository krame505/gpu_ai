#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>

#define CUDA_STACK_SIZE 1024 * 32

#define SEED 12345

#define BLOCK_SIZE 32

__global__ void simplePlayoutKernel(State *states, PlayerId *results, int n) {
  uint8_t tx = threadIdx.x;
  uint32_t bx = blockIdx.x;
  uint32_t tid = tx + (bx * NUM_LOCS);

  if (tid < n) {
    State state;
    state = states[tid];

    // Init random generator
    hiprandState_t generator;
    hiprand_init(SEED, tid, 0, &generator);
 
    bool gameOver = false;

    Loc playerOccupied[12];
    uint8_t numOccupied;
    bool noMoves[12];

    Move result[MAX_LOC_MOVES];
    
    do {
      numOccupied = 0;
      for (uint8_t i = 0; i < BOARD_SIZE; i++) {
	for (uint8_t j = 0; j < BOARD_SIZE; j++) {
	  Loc here(i, j);
	  if (state[here].occupied && state[here].owner == state.turn) {
	    playerOccupied[numOccupied] = here;
	    noMoves[numOccupied] = false;
	    numOccupied++;
	  }
	}
      }
      while (numOccupied > 0) {
	uint8_t tryLoc = (hiprand(&generator) % numOccupied) + 1;
	uint8_t index = 0;
	while (tryLoc > 0) {
	  while (noMoves[index] == true) {
	    index++;
	  }
	  tryLoc--;
	}
	uint8_t numMoves = state.genLocMoves(playerOccupied[index], result);
	if (numMoves == 0) {
	  numOccupied --;
	}
	else {
	  state.move(result[hiprand(&generator) % numMoves]);
	}
      }
      if (numOccupied == 0) {
	gameOver = true;
      }
    } while (!gameOver);

    results[tid] = state.getNextTurn();
  }
}

std::vector<PlayerId> DeviceSimplePlayoutDriver::runPlayouts(std::vector<State> states) const {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));

  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);

  hipError_t error = hipDeviceSetLimit(hipLimitStackSize, CUDA_STACK_SIZE);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error setting stack size: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  int numBlocks = states.size() / BLOCK_SIZE;
  if (states.size() % BLOCK_SIZE)
    numBlocks++;
  
  // Invoke the kernel
  simplePlayoutKernel<<<numBlocks, BLOCK_SIZE>>>(devStates, devResults, states.size());
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
