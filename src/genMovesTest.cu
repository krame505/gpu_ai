#include "hip/hip_runtime.h"

#include "genMovesTest.hpp"
#include "state.hpp"

#define CUDA_STACK_SIZE 1024 * 32

#include <iostream>
using namespace std;

__global__ void genMovesKernel(State *globalState, Move *globalMoves, uint8_t *globalNumMoves) {
  __shared__ State state;
  __shared__ Move moves[MAX_MOVES];

  state = *globalState;
  uint8_t numMoves = state.genMoves(moves);

  if (threadIdx.x == 0)
    *globalNumMoves = numMoves;

  for (unsigned i = 0; i < numMoves; i++) {
    unsigned index = i + threadIdx.x;
    globalMoves[index] = moves[index];
  }
}

bool genMovesTest(State state) {
  // Device variables
  State *devState;
  Move *devMoves;
  uint8_t *devNumMoves;
  hipMalloc(&devState, sizeof(State));
  hipMalloc(&devMoves, MAX_MOVES * sizeof(Move));
  hipMalloc(&devNumMoves, sizeof(uint8_t));

  // Copy states for playouts to device
  hipMemcpy(devState, &state, sizeof(State), hipMemcpyHostToDevice);

  hipError_t error = hipDeviceSetLimit(hipLimitStackSize, CUDA_STACK_SIZE);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error setting stack size: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Invoke the kernel
  genMovesKernel<<<1, NUM_LOCS>>>(devState, devMoves, devNumMoves);
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  Move gpuMoves[MAX_MOVES];
  hipMemcpy(gpuMoves, devMoves, MAX_MOVES * sizeof(Move), hipMemcpyDeviceToHost);
  
  uint8_t gpuNumMoves;
  hipMemcpy(&gpuNumMoves, devNumMoves, sizeof(uint8_t), hipMemcpyDeviceToHost);

  // Free the global memory
  hipFree(devState);
  hipFree(devMoves);
  hipFree(devNumMoves);

  Move cpuMoves[MAX_MOVES];
  uint8_t cpuNumMoves = state.genMoves(cpuMoves);

  bool match = true;

  if (cpuNumMoves != gpuNumMoves)
    match = false;

  for (uint8_t i = 0; i < cpuNumMoves; i++) {
    if (cpuMoves[i] != gpuMoves[i]) {
      match = false;
      break;
    }
  }

  if (!match) {
    cout << "Mismatch in CPU and GPU genMoves()" << endl;
    cout << state << endl;

    cout << "CPU Moves: " << (int)cpuNumMoves << endl;
    for (uint8_t i = 0; i < cpuNumMoves; i++) {
      cout << cpuMoves[i] << endl;
    }

    cout << "GPU Moves: " << (int)gpuNumMoves << endl;
    for (uint8_t i = 0; i < gpuNumMoves; i++) {
      cout << gpuMoves[i] << endl;
    }
  }

  return match;
}
