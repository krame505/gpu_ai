#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>

#define CUDA_STACK_SIZE 1024 * 32

#define SEED 12345

#define BLOCK_SIZE 32

__global__ void relaunchPlayoutKernel(State *states, PlayerId *results, int n, uint32_t *globaltempIndex) {
  uint8_t tx = threadIdx.x;
  uint32_t bx = blockIdx.x;
  uint32_t tid = tx + (bx * BLOCK_SIZE);
  uint32_t threadtempIndex = tid;


  if (tid < n) {
    State state;
    state = states[threadtempIndex];

    // Init random generator
    hiprandState_t generator;
    hiprand_init(SEED, threadtempIndex, 0, &generator);
 
    bool done = false;

    Move captureMoves[MAX_MOVES];
    Move directMoves[MAX_MOVES];
    uint8_t numMoveCapture, numMoveDirect;

    do {
      // Scan the board for pieces that can move
      numMoveCapture = 0;
      numMoveDirect = 0;
      for (uint8_t i = 0; i < BOARD_SIZE; i++) {
      	for (uint8_t j = 1 - (i % 2); j < BOARD_SIZE; j+=2) {
	        Loc here(i, j);
	        numMoveCapture += state.genLocSingleCaptureMoves(here, &captureMoves[numMoveCapture]);
	        numMoveDirect += state.genLocDirectMoves(here, &directMoves[numMoveDirect]);
	      }
      }

      if (numMoveCapture > 0) {
	      do {
	        uint8_t moveIndex = hiprand(&generator) % numMoveCapture;
	        Loc to = captureMoves[moveIndex].to;
	        state.move(captureMoves[moveIndex]);
	        state.turn = state.getNextTurn();
	        numMoveCapture = state.genLocSingleCaptureMoves(to, captureMoves);
	      } while (numMoveCapture > 0);
	        state.turn = state.getNextTurn();
      } 
      else if (numMoveDirect > 0) {
	      state.move(directMoves[hiprand(&generator) % numMoveDirect]);
      }
      else {
        results[threadtempIndex] = state.getNextTurn();
        if(*globaltempIndex >= n)
        	done = true;
        else{
          do {
          threadtempIndex = *globaltempIndex;
          } while (atomicCAS(globaltempIndex, threadtempIndex, *globaltempIndex + 1) != threadtempIndex);
           state = states[threadtempIndex]; 
        }
      }
    } while (!done);

  }
}

std::vector<PlayerId> DeviceRelaunchPlayoutDriver::runPlayouts(std::vector<State> states) {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));
 
  uint32_t *globaltempIndex;
  hipMalloc((void**) &globaltempIndex, sizeof(uint32_t));


  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);

  int numBlocks = 1024; //number of blocks  
  
  uint32_t temp = numBlocks * BLOCK_SIZE; //max number of threads that can run in parallel
  hipMemcpy(globaltempIndex, &temp, sizeof(uint32_t), hipMemcpyHostToDevice);

  hipError_t error;

  // Invoke the kernel
  relaunchPlayoutKernel<<<numBlocks, BLOCK_SIZE>>>(devStates, devResults, states.size(), globaltempIndex);
  hipDeviceSynchronize();
  

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Free the global memory
  hipFree(devStates);
  hipFree(devResults);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
