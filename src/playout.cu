#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>

#define CUDA_STACK_SIZE 1024 * 32

#define SEED 12345

__global__ void playoutKernel(State *states, PlayerId *results) {
  INIT_KERNEL_VARS

  __shared__ State state;
  state = states[blockIdx.x];

  // Init random generator
  hiprandState_t generator;
  hiprand_init(SEED, threadIdx.x + (blockIdx.x * NUM_LOCS), 0, &generator);
 
  // __shared__ uint8_t numDirectMoves[NUM_PLAYERS];
  // __shared__ uint8_t numCaptureMoves[NUM_PLAYERS];
  // __shared__ Move directMoves[NUM_PLAYERS][MAX_MOVES];
  // __shared__ Move captureMoves[NUM_PLAYERS][MAX_MOVES];
 
  __shared__ uint8_t numMoves[NUM_PLAYERS];
  __shared__ Move moves[NUM_PLAYERS][MAX_MOVES];

  __shared__ bool gameOver;

  if (threadIdx.x == 0)
    gameOver = false;

  do {
    state.genMoves(numMoves, moves);

    // Select a move
    // TODO: Optimize this portion
    if (id == 0) {
      Move move;
      if (numMoves[state.turn] > 0) {
        move = moves[state.turn][hiprand(&generator) % numMoves[state.turn]];
      }
      else {
        gameOver = true; // No moves, game is over
      }

      // Perform the move if there is one
      if (!gameOver)
	state.move(move);
    }
  } while (!gameOver);

  // TODO: Implement State::result to make use of parallelism
  if (threadIdx.x == 0)
    results[blockIdx.x] = state.result();
}

std::vector<PlayerId> DevicePlayoutDriver::runPlayouts(std::vector<State> states) const {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));

  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);

  hipError_t error = hipDeviceSetLimit(hipLimitStackSize, CUDA_STACK_SIZE);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error setting stack size: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Invoke the kernel
  playoutKernel<<<states.size(), NUM_LOCS>>>(devStates, devResults);
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
