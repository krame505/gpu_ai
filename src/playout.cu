#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>

#define CUDA_STACK_SIZE 1024 * 32

#define SEED 12345

__global__ void playoutKernel(State *states, PlayerId *results) {
  uint8_t tx = threadIdx.x;
  uint32_t bx = blockIdx.x;
  uint32_t tid = tx + (bx * NUM_LOCS);

  __shared__ State state;
  state = states[bx];

  // Init random generator
  hiprandState_t generator;
  hiprand_init(SEED, tid, 0, &generator);
 
  // __shared__ uint8_t numDirectMoves[NUM_PLAYERS];
  // __shared__ uint8_t numCaptureMoves[NUM_PLAYERS];
  // __shared__ Move directMoves[NUM_PLAYERS][MAX_MOVES];
  // __shared__ Move captureMoves[NUM_PLAYERS][MAX_MOVES];
 
  __shared__ Move moves[MAX_MOVES];

  __shared__ bool gameOver;

  if (tx == 0)
    gameOver = false;

  do {
    uint8_t numMoves = state.genMoves(moves);

    if (tx == 0) {
      // Select a move
      Move move;
      if (numMoves > 0) {
        move = moves[hiprand(&generator) % numMoves];
      }
      else {
        gameOver = true; // No moves, game is over
      }

      // Perform the move if there is one
      if (!gameOver)
	state.move(move);
    }
  } while (!gameOver);

  if (tx == 0)
    results[bx] = state.getNextTurn();
}

std::vector<PlayerId> DevicePlayoutDriver::runPlayouts(std::vector<State> states) const {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));

  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);

  hipError_t error = hipDeviceSetLimit(hipLimitStackSize, CUDA_STACK_SIZE);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error setting stack size: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Invoke the kernel
  playoutKernel<<<states.size(), NUM_LOCS>>>(devStates, devResults);
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Free the global memory
  hipFree(devStates);
  hipFree(devResults);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
