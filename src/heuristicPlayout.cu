#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"
#include "heuristic.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>
#include <cassert>

#define CUDA_STACK_SIZE 1024 * 32

#define SEED 12345

__global__ void heuristicPlayoutKernel(State *states, PlayerId *results) {
  uint8_t tx = threadIdx.x;
  uint32_t bx = blockIdx.x;
  uint32_t tid = tx + (bx * NUM_LOCS);

  __shared__ State state;
  state = states[bx];

  // Init random generator
  hiprandState_t generator;
  hiprand_init(SEED, tid, 0, &generator);
 
  __shared__ Move moves[MAX_MOVES];

  bool gameOver = false;

  do {
    uint8_t numMoves = state.genMovesParallel(moves);

    if (numMoves > 0) {
      Move optMove;
      float optScore = -1/0.0; // -infinity

      // Calculate scores for each move and copy the best ones for each thread into an array
      for (uint8_t i = tx; i < numMoves; i += NUM_LOCS) {
	Move move = moves[i];
	State newState = state;
	newState.move(move);
	float score = scoreHeuristic(newState) + hiprand_normal(&generator) * HEURISTIC_SIGMA;
	if (i == tx || score > optScore) {
	  optMove = move;
	  optScore = score;
	}
      }

      __shared__ float scores[NUM_LOCS];
      if (tx < numMoves) {
	moves[tx] = optMove;
	scores[tx] = optScore;
      }
      else {
	scores[tx] = -1/0.0; // -infinity
      }

      // Perform reduction to find move with max score
      for (uint8_t stride = NUM_LOCS / 2; stride > 0; stride >>= 1) {
	__syncthreads();
	if (tx < stride && scores[tx] < scores[tx + stride]) {
	  moves[tx] = moves[tx + stride];
	  scores[tx] = scores[tx + stride];
	}
      }

      if (tx == 0) {
	// Perform the move
	state.move(moves[0]);
      }
    }
    else {
      gameOver = true; // No moves, game is over
    }
  } while (!gameOver);

  if (tx == 0)
    results[bx] = state.getNextTurn();
}

std::vector<PlayerId> DeviceHeuristicPlayoutDriver::runPlayouts(std::vector<State> states) {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));

  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);

  // Increase default stack size
  hipError_t error = hipDeviceSetLimit(hipLimitStackSize, CUDA_STACK_SIZE);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error setting stack size: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Invoke the kernel
  heuristicPlayoutKernel<<<states.size(), NUM_LOCS>>>(devStates, devResults);
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Free the global memory
  hipFree(devStates);
  hipFree(devResults);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
