#include "hip/hip_runtime.h"

#include "playout.hpp"
#include "state.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>

#define SEED 12345

#define NUM_BLOCKS 192
#define BLOCK_SIZE 32

__global__ void coarsePlayoutKernel(State *states, PlayerId *results, size_t numStates, uint32_t *globalStateIndex) {
  uint8_t tx = threadIdx.x;
  uint32_t bx = blockIdx.x;
  uint32_t tid = tx + (bx * BLOCK_SIZE);

  uint32_t threadStateIndex = tid;

  if (tid < numStates) {
    State state;
    state = states[threadStateIndex];

    // Init random generator
    hiprandState_t generator;
    hiprand_init(SEED, threadStateIndex, 0, &generator);
 
    bool done = false;

    Move captureMoves[MAX_MOVES];
    Move directMoves[MAX_MOVES];
    uint8_t numMoveCapture, numMoveDirect;

    do {
      // Scan the board for pieces that can move
      numMoveCapture = 0;
      numMoveDirect = 0;
      for (uint8_t i = 0; i < BOARD_SIZE; i++) {
              for (uint8_t j = 1 - (i % 2); j < BOARD_SIZE; j+=2) {
          Loc here(i, j);
          numMoveCapture += state.genLocSingleCaptureMoves(here, &captureMoves[numMoveCapture]);
          numMoveDirect += state.genLocDirectMoves(here, &directMoves[numMoveDirect]);
        }
      }
      
      // Perform a random move if there is one
      if (numMoveCapture > 0) {
        do {
          uint8_t moveIndex = hiprand(&generator) % numMoveCapture;
          Loc to = captureMoves[moveIndex].to;
          state.move(captureMoves[moveIndex]);
          state.turn = state.getNextTurn();
          numMoveCapture = state.genLocSingleCaptureMoves(to, captureMoves);
        } while (numMoveCapture > 0);
        state.turn = state.getNextTurn();
      }
      else if (numMoveDirect > 0) {
        state.move(directMoves[hiprand(&generator) % numMoveDirect]);
      }
      else {
        // If the game is over, write the winner to the results array
        results[threadStateIndex] = state.getNextTurn();
        // Attempt to select another state from the inputs
        if (*globalStateIndex >= numStates)
          done = true;
        else {
          unsigned oldGlobalStateIndex;
          do {
            threadStateIndex = *globalStateIndex;
            oldGlobalStateIndex = atomicCAS(globalStateIndex, threadStateIndex, threadStateIndex + 1);
          } while (oldGlobalStateIndex != threadStateIndex);
          state = states[threadStateIndex]; 
        }
      }
    } while (!done);

  }
}

std::vector<PlayerId> DeviceCoarsePlayoutDriver::runPlayouts(std::vector<State> states) {
  // Device variables
  State *devStates;
  PlayerId *devResults;
  hipMalloc(&devStates, states.size() * sizeof(State));
  hipMalloc(&devResults, states.size() * sizeof(PlayerId));
 
  uint32_t *globalStateIndex;
  hipMalloc((void**) &globalStateIndex, sizeof(uint32_t));

  // Copy states for playouts to device
  hipMemcpy(devStates, states.data(), states.size() * sizeof(State), hipMemcpyHostToDevice);
  
  // Copy global state index to be the number of threads initially
  unsigned numThreads = NUM_BLOCKS * BLOCK_SIZE; // max number of threads that can run in parallel
  hipMemcpy(globalStateIndex, &numThreads, sizeof(uint32_t), hipMemcpyHostToDevice);

  hipError_t error;

  // Invoke the kernel
  coarsePlayoutKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(devStates, devResults, states.size(), globalStateIndex);
  hipDeviceSynchronize();

  // Check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    std::cout << "CUDA error calling kernel: " << hipGetErrorString(error) << std::endl;
    exit(1);
  }

  // Copy the results back to the host
  PlayerId results[states.size()];
  hipMemcpy(results, devResults, states.size() * sizeof(PlayerId), hipMemcpyDeviceToHost);

  // Free the global memory
  hipFree(devStates);
  hipFree(devResults);

  // Return a vector constructed from the contents of the array
  return std::vector<PlayerId>(results, results + states.size());
}
