#include "hip/hip_runtime.h"

#include "state.hpp"
#include "playout.hpp"

__global__ void playoutKernel(State *states, PlayerId *results) {
  uint8_t id = threadIdx.x;
  uint8_t row = id / (BOARD_SIZE / 2);
  uint8_t col = id % (BOARD_SIZE / 2) + (row % 2 == 0);
  Loc loc(row, col);

  __shared__ State state;
  state = states[blockIdx.x];

  __shared__ Move directMoves[NUM_PLAYERS][MAX_MOVES];
  __shared__ Move captureMoves[NUM_PLAYERS][MAX_MOVES];
  __shared__ uint8_t directMoveIndices[NUM_PLAYERS][NUM_LOCS];
  __shared__ uint8_t captureMoveIndices[NUM_PLAYERS][NUM_LOCS];

  while (!state.isFinished()) {
    PlayerId locOwner = state[loc].owner;

    // Generate direct and capture moves for this location
    Move locDirectMoves[MAX_LOC_MOVES];
    Move locCaptureMoves[MAX_LOC_MOVES];
    uint8_t numLocDirectMoves  = state.locDirectMoves(loc, locDirectMoves);
    uint8_t numLocCaptureMoves = state.locCaptureMoves(loc, locCaptureMoves);

    for (uint8_t i = 0; i < NUM_PLAYERS; i++) {
      if (locOwner == (PlayerId)i) {
	directMoveIndices[id][i]  = numLocDirectMoves;
	captureMoveIndices[id][i] = numLocCaptureMoves;
      }
      else {
	directMoveIndices[id][i]  = 0;
	captureMoveIndices[id][i] = 0;
      }
    }

    // Perform a reduction to calculate the max number of captures possible for each player
    for (unsigned i = NUM_LOCS / 2; i > 0; i >>= 1) {
      __syncthreads();
      if (i > id) {
	for (uint8_t i = 0; i < NUM_PLAYERS; i++) {
	  if (captureMoveIndices[i][id + i] > captureMoveIndices[i][id])
	    captureMoveIndices[i][id] = captureMoveIndices[i][id + i];
	}
      }
    }

    __shared__ uint8_t maxCaptureMoves[NUM_PLAYERS];
    if (id < NUM_PLAYERS)
      maxCaptureMoves[id] = captureMoveIndices[id][0];
    __syncthreads();

    // The number of capture moves for a location is 0 if there are any other locations with more capture moves
    for (uint8_t i = 0; i < NUM_PLAYERS; i++) {
      captureMoveIndices[i][id] = 0;
    }
    if (numLocCaptureMoves < maxCaptureMoves[locOwner])
      numLocCaptureMoves = 0;
    captureMoveIndices[locOwner][id] = numLocCaptureMoves;

    // Perform exclusive scans to get indices to copy moves into the shared arrays
    for (uint8_t stride = 1; stride <= NUM_LOCS; stride <<= 1) {
      __syncthreads();
      uint8_t i = (id + 1) * stride - 1; // TODO: Check that this is correct...
      if (i < NUM_LOCS) {
	for (uint8_t j = 0; j < NUM_PLAYERS; j++) {
	  directMoveIndices[j][i]  += directMoveIndices[j][i - stride];
	  captureMoveIndices[j][i] += captureMoveIndices[j][i - stride];
	}
      }
    }

    __syncthreads();
    if (id < NUM_PLAYERS) {
      directMoveIndices[id][NUM_LOCS - 1] = 0;
      captureMoveIndices[id][NUM_LOCS - 1] = 0;
    }

    for (uint8_t stride = NUM_LOCS / 2; stride > 0; stride >>= 1) {
      __syncthreads();
      int i = (id + 1) * stride - 1;
      uint8_t temp;
      for (uint8_t j = 0; j < NUM_PLAYERS; j++) {
	temp = directMoveIndices[j][i];
	directMoveIndices[j][i] += directMoveIndices[j][i - stride];
	directMoveIndices[j][i - stride] = temp;
	temp = captureMoveIndices[j][i];
	captureMoveIndices[j][i] += captureMoveIndices[j][i - stride];
	captureMoveIndices[j][i - stride] = temp;
      }
    }

    __syncthreads();

    // Copy generated moves to shared arrays
    for (uint8_t i = 0; i < numLocDirectMoves; i++) {
      //directMoves[i + directMoveIndices[id]] = locDirectMoves[i];
    }
  }
}

std::vector<PlayerId> playouts(std::vector<State>) {
  
}
