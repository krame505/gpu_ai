#include "hip/hip_runtime.h"

#include "genMovesTest.hpp"
#include "state.hpp"

__global__ void genMovesKernel(State *globalState, Move *globalMoves, uint8_t *globalNumMoves) {
  __shared__ State state;
  __shared__ uint8_t numMoves[NUM_PLAYERS];
  __shared__ Move moves[NUM_PLAYERS][MAX_MOVES];

  state = *globalState;
  state.genMoves(numMoves, moves);

  unsigned n, m;
  for (n = 0; n < NUM_PLAYERS; n ++) {
    globalNumMoves[n] = numMoves[n];
    for (m = 0; m < MAX_MOVES; m ++) {
      globalMoves[m + (n * MAX_MOVES)] = moves[n][m];
    }
  }
}

bool genMovesTest(State state) {
  // Device variables
  State *devState;
  Move *devMoves;
  uint8_t *devNumMoves;
  hipMalloc(&devState, sizeof(State));
  hipMalloc(&devMoves, NUM_PLAYERS * MAX_MOVES * sizeof(Move));
  hipMalloc(&devNumMoves, NUM_PLAYERS * sizeof(uint8_t));

  // Copy states for playouts to device
  hipMemcpy(devState, &state, sizeof(State), hipMemcpyHostToDevice);

  // Invoke the kernel
  genMovesKernel<<<1, 1>>>(devState, devMoves, devNumMoves);

  // Copy the results back to the host
  Move movesResult[NUM_PLAYERS * MAX_MOVES];
  hipMemcpy(movesResult, devMoves, NUM_PLAYERS * MAX_MOVES * sizeof(Move), hipMemcpyDeviceToHost);
  
  uint8_t numMovesResult[NUM_PLAYERS];
  hipMemcpy(numMovesResult, devNumMoves, NUM_PLAYERS * sizeof(uint8_t), hipMemcpyDeviceToHost);

  uint8_t numMoves[NUM_PLAYERS];
  Move result[NUM_PLAYERS][MAX_MOVES];
  state.genMoves(numMoves, result);

  // TODO: Compare the results

  return true;
}
